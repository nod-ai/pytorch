#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/native/UnaryOps.h>
#include <ATen/zoom/jit/Loops.cuh>
#include <ATen/zoom/jit/JitLoops.cuh>
#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/zoom/Math.cuh>
#include <c10/util/TypeSafeSignMath.h>
#include <ATen/OpMathType.h>

#include <type_traits>

namespace at::native {

void logical_not_kernel_zoom(TensorIteratorBase& iter) {
  // error check -- this is just ensuring we don't dispatch on types that aren't in ALL_TYPES_AND_COMPLEX_AND3(...)
  // so we don't have to maintain a separate list or to do double dispatch.
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(kBool, kHalf, kBFloat16, iter.dtype(0), "logical_not_zoom", [&]() {});

  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(kBool, kHalf, kBFloat16, iter.dtype(1), "logical_not_zoom", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> bool { return !a; });
  });
}

// NB: Ignores the negative bit on tensors
CONSTEXPR_EXCEPT_WIN_CUDA char neg_name[] = "neg_kernel";
void neg_kernel_zoom(TensorIteratorBase& iter) {
  auto dtype = iter.dtype();
  if (at::isComplexType(dtype)) {
  static const auto neg_string = jiterator_stringify(
      template <typename T>
      T neg_kernel(T a) {
        return -a;
      }
  ); // neg_string
  AT_DISPATCH_COMPLEX_TYPES_AND(kComplexHalf, dtype, "neg_zoom", [&]() {
      jitted_gpu_kernel<
        /*name=*/ neg_name,
        /*return_dtype=*/ scalar_t,
        /*common_dtype=*/ scalar_t,
        /*arity=*/ 1>(iter, neg_string);
  });

  } else {
  AT_DISPATCH_ALL_TYPES_AND2(ScalarType::Half, ScalarType::BFloat16, dtype, "neg_zoom", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return -a;
    });
  });
  }
}

void sign_kernel_zoom(TensorIteratorBase& iter){
  if (iter.dtype() == ScalarType::Bool) {
    gpu_kernel(iter, []GPU_LAMBDA(bool a){
      return a;
    });
  } else {
    AT_DISPATCH_ALL_TYPES_AND2(ScalarType::Half, ScalarType::BFloat16, iter.dtype(), "sign_zoom", [&]() {
        gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
            return c10::signum(a);
        });
    });
  }
}

void signbit_kernel_zoom(TensorIteratorBase& iter){
  // NOTE: signbit does not always support integral arguments.
  if (at::isIntegralType(iter.input_dtype(), /*includeBool=*/false)) {
    AT_DISPATCH_INTEGRAL_TYPES(iter.input_dtype(), "signbit_zoom", [&]() {
      gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> bool { return is_negative(a); });
    });
  } else {
    AT_DISPATCH_FLOATING_TYPES_AND2(kBFloat16, ScalarType::Half, iter.input_dtype(), "signbit_zoom", [&]() {
      using opmath_t = at::opmath_type<scalar_t>;
      gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> bool { return signbit(opmath_t{a}); });
    });
  }
}

template<typename T>
C10_HOST_DEVICE static inline c10::complex<T> sgn_wrapper(c10::complex<T> z) {
  if (z == c10::complex<T>(0, 0)) {
    return c10::complex<T>(0, 0);
  } else {
    return z / std::abs(z);
  }
}

CONSTEXPR_EXCEPT_WIN_CUDA char sgn_name[] = "sgn_kernel";
void sgn_kernel_zoom(TensorIteratorBase& iter){
  auto dtype = iter.dtype();
    static const auto sgn_string = jiterator_stringify(
        template <typename T>
        T sgn_kernel(T z) {
          const T zero = T(0);
          if (z == zero) {
            return zero;
          } else {
            return z / std::abs(z);
          }
        }
      ); // sgn_string
    AT_DISPATCH_COMPLEX_TYPES_AND(kComplexHalf, dtype, "sgn_zoom", [&]() {
      jitted_gpu_kernel<
        /*name=*/ sgn_name,
        /*return_dtype=*/ scalar_t,
        /*common_dtype=*/ scalar_t,
        /*arity=*/ 1>(iter, sgn_string);
      });
}

REGISTER_PRIVATEUSE1_DISPATCH(logical_not_stub, &logical_not_kernel_zoom);
REGISTER_PRIVATEUSE1_DISPATCH(neg_stub, &neg_kernel_zoom);
REGISTER_PRIVATEUSE1_DISPATCH(sign_stub, &sign_kernel_zoom);
REGISTER_PRIVATEUSE1_DISPATCH(signbit_stub, &signbit_kernel_zoom);
REGISTER_PRIVATEUSE1_DISPATCH(sgn_stub, &sgn_kernel_zoom);

} // namespace at::native