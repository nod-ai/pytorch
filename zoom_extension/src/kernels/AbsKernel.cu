#include "hip/hip_runtime.h"
// #define TORCH_ASSERT_NO_OPERATORS
#include <ATen/native/UnaryOps.h>
#include "../jit/Loops.cuh"
#include "../jit/JitLoops.cuh"
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include "../extension.h"
// #include <torch/library.h>

namespace at::native {


/*CONSTEXPR_EXCEPT_WIN_CUDA*/ constexpr char abs_name[] = "abs_kernel";
void abs_kernel_zoom(TensorIteratorBase& iter) {
  auto dtype = iter.dtype();
  static const auto abs_string = jiterator_stringify(
        template <typename T> T abs_kernel(T x) { return std::abs(x); });
  if (at::isComplexType(dtype)) {
    AT_DISPATCH_COMPLEX_TYPES_AND(kComplexHalf, dtype, "abs_zoom", [&]() {
      jitted_gpu_kernel<
          /*name=*/abs_name,
          /*return_dtype=*/scalar_t,
          /*common_dtype=*/scalar_t,
          /*arity=*/1>(iter, abs_string);
    });
  } else {
    AT_DISPATCH_ALL_TYPES_AND3(ScalarType::Half,
        ScalarType::BFloat16,
        ScalarType::Bool,
        iter.dtype(),
        "abs_zoom", [&]() {
      jitted_gpu_kernel<
          /*name=*/abs_name,
          /*return_dtype=*/scalar_t,
          /*common_dtype=*/scalar_t,
          /*arity=*/1>(iter, abs_string);
    });
  }
}

REGISTER_PRIVATEUSE1_DISPATCH(abs_stub, &abs_kernel_zoom);

// Tensor& abs_out(const Tensor& self, Tensor& result) {
//   return unary_op_impl_with_complex_to_float_out(result, self, abs_stub, /*promotes_integer_to_float=*/false);
// }

// Tensor abs(const Tensor & self) {
//   return unary_op_impl_with_complex_to_float(self, at::zoom::native::abs_out);
// }

TORCH_LIBRARY_IMPL(aten, PrivateUse1, m) {
  m.impl("abs", &abs);
  m.impl("abs.out", &abs_out);
}


} // namespace at::native